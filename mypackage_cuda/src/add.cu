#include "hip/hip_runtime.h"
#include "add.h"

__global__ void add_kernel(const float* __restrict__ a,
                           const float* __restrict__ b, float* __restrict__ out,
                           int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] + b[i];
  }
}

void add_cpp(uintptr_t prt_a, uintptr_t prt_b, uintptr_t prt_result, int n) {
  printf("[CPP LOG] Enter add_cpp().\n");

  const float* a = reinterpret_cast<const float*>(prt_a);
  const float* b = reinterpret_cast<const float*>(prt_b);
  float* result = reinterpret_cast<float*>(prt_result);

  // for (int i = 0; i < n; i++) {
  //   result[i] = a[i] + b[i];
  // }

  // Simple launch config
  const int threads = 256;
  const int blocks = (n + threads - 1) / threads;

  hipStream_t stream = 0;
  add_kernel<<<blocks, threads, 0, stream>>>(a, b, result, n);

  // Optional: surface launch errors immediately
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "[CUDA ERR] Kernel launch failed: %s\n",
            hipGetErrorString(err));
    return;
  }

  // Optional: sync here if you need completion before returning.
  // If you prefer async behavior, remove this and let the caller sync on the
  // stream.
  err = hipStreamSynchronize(stream);
  if (err != hipSuccess) {
    fprintf(stderr, "[CUDA ERR] Stream sync failed: %s\n",
            hipGetErrorString(err));
  }
}
