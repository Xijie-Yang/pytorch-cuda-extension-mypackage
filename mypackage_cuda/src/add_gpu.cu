#include "hip/hip_runtime.h"
#include "add_gpu.h"

__global__ void add_kernel(const float* __restrict__ a,
                           const float* __restrict__ b, float* __restrict__ out,
                           int n) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    out[i] = a[i] + b[i];
  }
}

void add_cuda(uintptr_t prt_a, uintptr_t prt_b, uintptr_t prt_result, int n) {
  printf("[C LOG] Enter add_cuda().\n");

  const float* a = reinterpret_cast<const float*>(prt_a);
  const float* b = reinterpret_cast<const float*>(prt_b);
  float* result = reinterpret_cast<float*>(prt_result);

  // Simple launch config
  const int threads = 256;
  const int blocks = (n + threads - 1) / threads;

  add_kernel<<<blocks, threads>>>(a, b, result, n);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
}
